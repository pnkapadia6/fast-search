#include "hip/hip_runtime.h"
// Parallel implementation of string searching
// a?b gives -> acb, abb, ...
// case insensitive
// using local memory
// blocks & threads

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstdio>
#include <math.h>

using namespace std;

/* M - window size */
#define M 20    

__global__ void searchKeywordKernel(int *result, char *data, char *keyword,int keyword_len)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x , j;
        result[i]=0;
        char * s= (char *) malloc (M + keyword_len -1);
        for (j = 0; j < M + keyword_len - 1; j++)
        {
                s[j] = data[j + (i * M)];
        }
        s[j]='\0';
        keyword[keyword_len]='\0';
//      printf("TT Keyword=%s   %d\n",keyword, keyword_len);
//      printf("Line %d -> %s  M = %d\n", i, s, M);
        bool flag=0;
        int k=0;
        for (int j = 0; j <= M; j++)
        {
                if(s[j] >= 65 && s[j] <= 90)
                        s[j]+=32;
                if (s[j] == keyword[0] || keyword[0]=='?')
                {
                        flag=0;
                        for (k = 1; k < keyword_len; k++)
                        {
                                if(keyword[k]=='?')
                                        continue;
                                if(s[k+j] >= 65 && s[k+j] <= 90)
                                        s[k+j] += 32;
                                if (s[k + j] != keyword[k] || s[k + j]==' ' || s[k + j]=='\n')
                                {
                                        flag=0;
                                        break;
                                }
                                else
                                {
                                        flag=1;
                                }
                        }

                        if(flag==1)
                                result[i]=result[i]+1;

                }
        }
        __syncthreads();
}

int main()
{
        hipEvent_t k_start, k_stop, t_start, t_stop, c_start, c_stop;
        hipEventCreate(&t_start);
        hipEventCreate(&t_stop);
        hipEventRecord(t_start, 0);

        hipEventCreate(&c_start);
        hipEventCreate(&c_stop);
        hipEventRecord(c_start, 0);

        std::ifstream t("text_150.txt");
        std::stringstream buffer;
        buffer << t.rdbuf();

        string data_s = buffer.str();
        const char *data = data_s.c_str();

        hipEventRecord(c_stop, 0);
        hipEventSynchronize(c_stop);
        float copy_time;
        hipEventElapsedTime(&copy_time, c_start, c_stop);
        printf("\nCopy time: %f msec\n",copy_time);

//      printf("\nM=%d\n", M);
//      printf("Data size = %ld \n",data_s.size());
//      printf("%s\n",data);

        t.close();

        int num_blocks = ceil(data_s.size()/(float)(1024 * M)) ;
        int num_threads = ceil(data_s.size()/(float)(M*num_blocks));

//      printf("No of threads = %d  blocks=%d  \n",num_threads, num_blocks);
        char *keyword = "ab";
        size_t keyword_len = strlen(keyword);

//      printf("Keyword=%s   %ld \n",keyword, keyword_len);

        int *result = (int *) malloc(num_blocks * num_threads * sizeof(int));
        memset(result, 0, num_blocks * num_threads);

        //device data
        char *dev_data = 0;
        char *dev_keyword = 0;
        int *dev_result = 0;

        // Allocate GPU buffers for result set.
        hipMalloc((void**) &dev_result, num_blocks * num_threads * sizeof(int));
        hipMalloc((void**) &dev_data, data_s.size() + 1);
        hipMalloc((void**) &dev_keyword, keyword_len);

        // Copy input data and keyword from host memory to GPU buffers.
        hipMemcpy(dev_data, data, data_s.size() + 1, hipMemcpyHostToDevice);
        hipMemcpy(dev_keyword, keyword, keyword_len, hipMemcpyHostToDevice);
        hipMemcpy(dev_result, result, num_blocks * num_threads, hipMemcpyHostToDevice);

        hipEventCreate(&k_start);
        hipEventCreate(&k_stop);
        hipEventRecord(k_start, 0);

        // Launch a search keyword kernel on the GPU with one thread for each element.
        searchKeywordKernel<<<num_blocks, num_threads>>>(dev_result, dev_data, dev_keyword, keyword_len);
        hipDeviceSynchronize();

        // Copy result from GPU buffer to host memory.
        hipMemcpy(result, dev_result, num_blocks * num_threads * sizeof(int),hipMemcpyDeviceToHost);

        hipEventRecord(k_stop, 0);
        hipEventSynchronize(k_stop);
        float kernel_time;
        hipEventElapsedTime(&kernel_time, k_start, k_stop);
        printf("\nKernel time: %f msec\n",kernel_time);


        printf("\n");
        int total_matches = 0;
        for (int i = 0; i < num_threads * num_blocks; i++)
        {
                if (result[i] > 0)
                {
                //      printf("%d matches found at line %d \n",result[i], i);
                        total_matches=total_matches+result[i];
                }
        }
        printf("Total matches = %d\n", total_matches);
        hipFree(dev_result);
        hipFree(dev_data);
        hipFree(dev_keyword);


        hipEventRecord(t_stop, 0);
        hipEventSynchronize(t_stop);
        float total_time;
        hipEventElapsedTime(&total_time, t_start, t_stop);
        printf("\nTotal time: %f msec\n",total_time);

        return 0;
}